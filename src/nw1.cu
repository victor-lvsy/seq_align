#include "hip/hip_runtime.h"
#include "nw1.cuh"

// Kernel for initializing borders of the score matrix
__global__ void init_borders(int *d_score, int n, int m, int gap) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx <= n) {
        d_score[idx * (m + 1)] = idx * gap; // Initialize first column
    }
    if (idx <= m) {
        d_score[idx] = idx * gap; // Initialize first row
    }
}

// Kernel for filling the matrix using the anti-diagonal approach
__global__ void fill_matrix(int *d_score, const char *d_seq1, const char *d_seq2, int match, int mismatch, int gap, int n, int m) {
    int total_diagonals = n + m - 1;
    int tid = threadIdx.x;
    
    for (int diag = 2; diag <= total_diagonals; ++diag) {
        int i = diag - tid;
        int j = tid;
        if (i >= 1 && i <= n && j >= 1 && j <= m) {
            int idx = i * (m + 1) + j;
            int match_score = (d_seq1[i - 1] == d_seq2[j - 1]) ? match : mismatch;
            d_score[idx] = max(
                d_score[(i - 1) * (m + 1) + (j - 1)] + match_score,
                max(
                    d_score[(i - 1) * (m + 1) + j] + gap,
                    d_score[i * (m + 1) + (j - 1)] + gap
                )
            );
        }
    }
}

void nw1(const std::string &seq1, const std::string &seq2, int match, int mismatch, int gap) {
    printf("HELLO THERE, i am nw1\n");
    int n = seq1.length();
    int m = seq2.length();

    // Allocating matrix with padding
    int *h_score = new int[(n + 1) * (m + 1)];
    std::fill(h_score, h_score + (n + 1) * (m + 1), 0);
    
    int *d_score;
    char *d_seq1, *d_seq2;
    
    // Allocate memory on GPU
    hipMalloc(&d_score, (n + 1) * (m + 1) * sizeof(int));
    hipMalloc(&d_seq1, n * sizeof(char));
    hipMalloc(&d_seq2, m * sizeof(char));
    
    // Copy sequences to GPU
    hipMemcpy(d_seq1, seq1.data(), n * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_seq2, seq2.data(), m * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_score, h_score, (n + 1) * (m + 1) * sizeof(int), hipMemcpyHostToDevice);
    
    int num_threads = max(n, m) + 1;
    int num_blocks = (num_threads + (NUMBER_OF_THREADS - 1)) / NUMBER_OF_THREADS;
    
    // Initialize borders of the matrix on GPU
    init_borders<<<num_blocks, NUMBER_OF_THREADS>>>(d_score, n, m, gap);
    
    // Fill the matrix on GPU
    fill_matrix<<<num_blocks, NUMBER_OF_THREADS>>>(d_score, d_seq1, d_seq2, match, mismatch, gap, n, m);
    
    // Copy score matrix back to CPU
    hipMemcpy(h_score, d_score, (n + 1) * (m + 1) * sizeof(int), hipMemcpyDeviceToHost);
    
    // Clean up GPU memory
    hipFree(d_score);
    hipFree(d_seq1);
    hipFree(d_seq2);

    // Print score matrix for debugging
    // for (int i = 0; i <= n; ++i) {
    //     for (int j = 0; j <= m; ++j) {
    //         std::cout << h_score[i * (m + 1) + j] << " ";
    //     }
    //     std::cout << std::endl;
    // }

    // Backtracking
    int i = n;
    int j = m;
    std::string aligned_seq1, aligned_seq2;

    while (i > 0 && j > 0) {
        if (h_score[i * (m + 1) + j] == h_score[(i - 1) * (m + 1) + (j - 1)] + (seq1[i - 1] == seq2[j - 1] ? match : mismatch)) {
            aligned_seq1 = seq1[i - 1] + aligned_seq1;
            aligned_seq2 = seq2[j - 1] + aligned_seq2;
            --i;
            --j;
        } else if (h_score[i * (m + 1) + j] == h_score[(i - 1) * (m + 1) + j] + gap) {
            aligned_seq1 = seq1[i - 1] + aligned_seq1;
            aligned_seq2 = "-" + aligned_seq2;
            --i;
        } else {
            aligned_seq1 = "-" + aligned_seq1;
            aligned_seq2 = seq2[j - 1] + aligned_seq2;
            --j;
        }
    }

    while (i > 0) {
        aligned_seq1 = seq1[i - 1] + aligned_seq1;
        aligned_seq2 = "-" + aligned_seq2;
        --i;
    }

    while (j > 0) {
        aligned_seq1 = "-" + aligned_seq1;
        aligned_seq2 = seq2[j - 1] + aligned_seq2;
        --j;
    }

    std::cout << "Aligned Sequence 1: " << aligned_seq1 << std::endl;
    std::cout << "Aligned Sequence 2: " << aligned_seq2 << std::endl;

    delete[] h_score;
}
