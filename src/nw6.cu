#include "hip/hip_runtime.h"
#include "nw6.cuh"

// Dynamic kernel launches to maximize gpu usage. 
// Disclaimer: This is a simplifed version of the algorithm because longer sequences are split 
// and processed without codependances.

__global__ void kernel_launcher(int *d_score, char *d_seq1, char *d_seq2, int match, int mismatch, int gap, int n, int m){
    int tid = threadIdx.x, number_of_subthreads = NUMBER_OF_THREADS;
    int *test = d_score + tid * (number_of_subthreads * number_of_subthreads);
    char *seq1 = d_seq1 + tid * number_of_subthreads;
    char *seq2 = d_seq2 + tid * number_of_subthreads;

    init_borders_v6<<<1, number_of_subthreads>>>(test, number_of_subthreads, gap);
    fill_matrix_v6<<<1, number_of_subthreads>>>(test, seq1, seq2, match, mismatch, gap, number_of_subthreads - 1, number_of_subthreads - 1);
}

// Kernel for initializing borders of the score matrix
__global__ void init_borders_v6(int *d_score, int n, int gap)
{
    int idx = threadIdx.x;
    int total_diagonals = n;
    
    // Initialize diagonals for the border elements
    if (idx < total_diagonals) {
        if (idx <= n) {
            // Initialize first column elements
            int diag_index = idx;
            int diag_position = 0;
            int pos = diag_index * (diag_index + 1) / 2 + diag_position;
            d_score[pos] = idx * gap; // Setting gap penalties for the first column
        }

        if (idx <= n) {
            // Initialize first row elements
            int diag_index = idx;
            int diag_position = idx;
            int pos = diag_index * (diag_index + 1) / 2 + diag_position;
            d_score[pos] = idx * gap; // Setting gap penalties for the first row
        }
    }
}

// Kernel for filling the matrix using the anti-diagonal approach
__global__ void fill_matrix_v6(int *d_score, const char *d_seq1, const char *d_seq2, int match, int mismatch, int gap, int n, int m) {
    const int total_diagonals = n + m - 1;
    const int tid = threadIdx.x;
    int first_i, last_i, elements_in_diag, i, j;
    int idx = 4, mem = 2, mem2 = 1;
    
    for (int diag = 2; diag <= total_diagonals + 1; ++diag) {
        first_i = max(1, diag - m); // Calculate start i index for the current diagonal
        last_i = min(n, diag - 1);  // Calculate end i index for the current diagonal
        elements_in_diag = last_i - first_i + 1; // Number of elements in the current diagonal

        if (tid < elements_in_diag) {
            i = last_i - tid; // Mapping tid to i
            j = diag - i; // Mapping i to j

            // Create the linear indexes related to the antidiagonal memory accesses.
            int linear_index = idx + tid;
            int linear_index_l = mem + tid;
            int linear_index_t = (diag <= n) ? linear_index_l - 1 : linear_index_l + 1;
            int linear_index_tl = (diag <= n + 1) ? mem2 + tid - 1 : ((diag == n + 2) ? mem2 + tid : mem2 + tid + 1 );
            if (linear_index <= (n + 1) * (m + 1)) {
                int match_score = (d_seq1[i - 1] == d_seq2[j - 1]) ? match : mismatch;
                d_score[linear_index] = max(
                    d_score[linear_index_tl] + match_score,
                    max(
                        d_score[linear_index_l] + gap,
                        d_score[linear_index_t] + gap));
            }
        }
        mem2 = mem;
        mem = idx;
        idx += elements_in_diag;
        if(diag < n){
            idx+=2;
        }
        if(diag == n){
            idx+=1;
        }
        __syncthreads();
    }
}

// Function to convert a diagonal-major format matrix to row-major format
void convertDiagonalToRowMajor3(int* diagMajorMatrix, int n, int* rowMajorMatrix) {
    n -= 1;
    int total_diagonals = 2*n - 1;
    int idx = 0;
    for (int diag = 0; diag <= total_diagonals + 1; ++diag) {
        int first_i = max(0, diag - n); // Calculate start i index for the current diagonal
        int last_i = min(n, diag);  // Calculate end i index for the current diagonal
        int elements_in_diag = last_i - first_i; // Number of elements in the current diagonal
        if(diag > n){
            for(int i = 0; i <= elements_in_diag; i++){
                rowMajorMatrix[(n - i) * (n+1) + (diag - (n - i))] = diagMajorMatrix[idx + i];
            }
        }
        else{
            for(int i = 0; i <= elements_in_diag; i++){
                rowMajorMatrix[(i * (n+1)) + (elements_in_diag - i)] = diagMajorMatrix[idx + i];
            }
        }
        
        idx += elements_in_diag + 1;

    }
}


// Host function for Needleman-Wunsch algorithm
void nw6(const std::string &seq1, const std::string &seq2, int match, int mismatch, int gap)
{
    printf("HELLO THERE, I am nw4\n");

    int n = seq1.length();
    int m = seq2.length();

    printf("Size of seq1: %d\nSize of seq2: %d\nProduct: %d\n", n, m, n * m);

    // Allocate the matrix with padding (extra row and column) on the host
    int *h_score = new int[(n + 1) * (m + 1)];
    int *h_score_r = new int[(n + 1) * (m + 1)];
    std::fill(h_score, h_score + (n + 1) * (m + 1), 0); // Initialize the matrix with zeros

    int *d_score;
    char *d_seq1, *d_seq2;

    // Allocate memory on GPU for score matrix and sequences
    CHECK(hipMalloc(&d_score, (n + 1) * (m + 1) * sizeof(int)));
    CHECK(hipMalloc(&d_seq1, n * sizeof(char)));
    CHECK(hipMalloc(&d_seq2, m * sizeof(char)));

    // Copy sequences from host to GPU
    CHECK(hipMemcpy(d_seq1, seq1.data(), n * sizeof(char), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_seq2, seq2.data(), m * sizeof(char), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_score, h_score, (n + 1) * (m + 1) * sizeof(int), hipMemcpyHostToDevice));

    hipStream_t stream1, stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

    int num_threads = (n + (NUMBER_OF_THREADS - 1)) / NUMBER_OF_THREADS;
    printf("Number of Threads: %d\n", num_threads);
    kernel_launcher<<<1, num_threads,0,stream1>>>(d_score, d_seq1, d_seq2, match, mismatch, gap, n, m);
    kernel_launcher<<<1, num_threads,0,stream2>>>(d_score, d_seq1, d_seq2, match, mismatch, gap, n, m);
    CHECK_KERNELCALL();
    CHECK(hipDeviceSynchronize());
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);


    // Initialize borders of the matrix on GPU
    // int num_threads = max(n, m);
    // int num_blocks = (num_threads + (NUMBER_OF_THREADS - 1)) / NUMBER_OF_THREADS;
    // printf("Number of blocks: %d\nNumber of Threads: %d\n", num_blocks, num_threads);
    // init_borders_v6<<<num_blocks, NUMBER_OF_THREADS>>>(d_score, n, m, gap);
    // CHECK_KERNELCALL();
    // CHECK(hipDeviceSynchronize());


    // Fill the matrix on GPU
    // fill_matrix_v6<<<num_blocks, NUMBER_OF_THREADS>>>(d_score, d_seq1, d_seq2, match, mismatch, gap, n, m);
    // CHECK_KERNELCALL();
    // CHECK(hipDeviceSynchronize());
        

    // Copy score matrix back to CPU from GPU
    CHECK(hipMemcpy(h_score, d_score, (n + 1) * (m + 1) * sizeof(int), hipMemcpyDeviceToHost));

    // Clean up GPU memory
    CHECK(hipFree(d_score));
    CHECK(hipFree(d_seq1));
    CHECK(hipFree(d_seq2));

    // Print score matrix for debugging
    // for (int i = 0; i <= (m+1)*(n+1); ++i) {
    //     std::cout << h_score[i] << " ";
    // }

    // int *h_score2 = h_score + NUMBER_OF_THREADS*NUMBER_OF_THREADS;

    // Print score matrix for debugging
    // convertDiagonalToRowMajor3(h_score,NUMBER_OF_THREADS,h_score_r);

    // std::cout << std::endl;
    // std::cout << std::endl;

    // Print the row-major matrix
    // for (int i = 0; i < NUMBER_OF_THREADS; ++i) {
    //     for (int j = 0; j < NUMBER_OF_THREADS; ++j) {
    //         std::cout << h_score_r[i * (NUMBER_OF_THREADS) + j] << " ";
    //     }
    //     std::cout << std::endl;
    // }

    // std::cout << std::endl;
    // std::cout << std::endl;

    // convertDiagonalToRowMajor3(h_score2,NUMBER_OF_THREADS,h_score_r);

    // for (int i = 0; i < NUMBER_OF_THREADS; ++i) {
    //     for (int j = 0; j < NUMBER_OF_THREADS; ++j) {
    //         std::cout << h_score_r[i * (NUMBER_OF_THREADS) + j] << " ";
    //     }
    //     std::cout << std::endl;
    // }

    // Backtracking
    // int i = n;
    // int j = m;
    // std::string aligned_seq1, aligned_seq2;

    // while (i > 0 && j > 0) {
    //     if (h_score_r[i * (m + 1) + j] == h_score_r[(i - 1) * (m + 1) + (j - 1)] + (seq1[i - 1] == seq2[j - 1] ? match : mismatch)) {
    //         aligned_seq1 = seq1[i - 1] + aligned_seq1;
    //         aligned_seq2 = seq2[j - 1] + aligned_seq2;
    //         --i;
    //         --j;
    //     } else if (h_score_r[i * (m + 1) + j] == h_score_r[(i - 1) * (m + 1) + j] + gap) {
    //         aligned_seq1 = seq1[i - 1] + aligned_seq1;
    //         aligned_seq2 = "-" + aligned_seq2;
    //         --i;
    //     } else {
    //         aligned_seq1 = "-" + aligned_seq1;
    //         aligned_seq2 = seq2[j - 1] + aligned_seq2;
    //         --j;
    //     }
    // }

    // while (i > 0) {
    //     aligned_seq1 = seq1[i - 1] + aligned_seq1;
    //     aligned_seq2 = "-" + aligned_seq2;
    //     --i;
    // }

    // while (j > 0) {
    //     aligned_seq1 = "-" + aligned_seq1;
    //     aligned_seq2 = seq2[j - 1] + aligned_seq2;
    //     --j;
    // }

    // std::cout << "Aligned Sequence 1: " << aligned_seq1 << std::endl;
    // std::cout << "Aligned Sequence 2: " << aligned_seq2 << std::endl;

    delete[] h_score;
    delete[] h_score_r;
}
